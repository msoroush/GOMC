
#include <hip/hip_runtime.h>
/*******************************************************************************
GPU OPTIMIZED MONTE CARLO (GOMC) 2.50
Copyright (C) 2018  GOMC Group
A copy of the GNU General Public License can be found in the COPYRIGHT.txt
along with this program, also can be found at <http://www.gnu.org/licenses/>.
********************************************************************************/
#ifdef GOMC_CUDA

#include <cuda.h>
#include "CalculateForceCUDAKernel.cuh"
#include "CalculateEnergyCUDAKernel.cuh"
#include "ConstantDefinitionsCUDAKernel.cuh"
#include "CalculateMinImageCUDAKernel.cuh"
#include "cub/cub.cuh"
#include <stdio.h>

using namespace cub;
#define NUMBER_OF_NEIGHBOR_CELL 27

void CallBoxInterForceGPU(VariablesCUDA *vars,
                          vector<int> &cellVector,
                          vector<int> &cellStartIndex,
                          std::vector<std::vector<int> > &neighborList,
                          vector<int> &mapParticleToCell,
                          XYZArray const &currentCoords,
                          XYZArray const &currentCOM,
                          BoxDimensions const &boxAxes,
                          bool electrostatic,
                          vector<double> &particleCharge,
                          vector<int> &particleKind,
                          vector<int> &particleMol,
                          double &rT11,
                          double &rT12,
                          double &rT13,
                          double &rT22,
                          double &rT23,
                          double &rT33,
                          double &vT11,
                          double &vT12,
                          double &vT13,
                          double &vT22,
                          double &vT23,
                          double &vT33,
                          bool sc_coul,
                          double sc_sigma_6,
                          double sc_alpha,
                          uint sc_power,
                          uint const box)
{
  int atomNumber = currentCoords.Count();
  int molNumber = currentCOM.Count();
  int neighborListCount = neighborList.size() * NUMBER_OF_NEIGHBOR_CELL;
  int numberOfCells = neighborList.size();
  int *gpu_particleKind;
  int *gpu_particleMol;
  int *gpu_neighborList, *gpu_cellStartIndex;
  int blocksPerGrid, threadsPerBlock;
  double *gpu_particleCharge;
  double *gpu_final_value;

  // Run the kernel...
  threadsPerBlock = 256;
  blocksPerGrid = (int)(numberOfCells * NUMBER_OF_NEIGHBOR_CELL);

  // Convert neighbor list to 1D array
  std::vector<int> neighborlist1D(neighborListCount);
  for(int i=0; i<neighborList.size(); i++) {
    for(int j=0; j<NUMBER_OF_NEIGHBOR_CELL; j++) {
      neighborlist1D[i*NUMBER_OF_NEIGHBOR_CELL + j] = neighborList[i][j];
    }
  }

  gpuErrchk(cudaMalloc((void**) &gpu_neighborList, neighborListCount * sizeof(int)));
  gpuErrchk(cudaMalloc((void**) &gpu_cellStartIndex,
                       cellStartIndex.size() * sizeof(int)));
  cudaMalloc((void**) &gpu_particleCharge,
             particleCharge.size() * sizeof(double));
  cudaMalloc((void**) &gpu_particleKind, particleKind.size() * sizeof(int));
  cudaMalloc((void**) &gpu_particleMol, particleMol.size() * sizeof(int));
  cudaMalloc((void**) &gpu_final_value, sizeof(double));
  cudaMalloc(&vars->gpu_rT11, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
             threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_rT12, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
             threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_rT13, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
             threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_rT22, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
             threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_rT23, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
             threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_rT33, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
             threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_vT11, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
             threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_vT12, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
             threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_vT13, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
             threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_vT22, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
             threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_vT23, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
             threadsPerBlock * sizeof(double));
  cudaMalloc(&vars->gpu_vT33, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
             threadsPerBlock * sizeof(double));

  gpuErrchk(cudaMemcpy(vars->gpu_mapParticleToCell, &mapParticleToCell[0],
                       atomNumber * sizeof(int), cudaMemcpyHostToDevice));
  gpuErrchk(cudaMemcpy(gpu_neighborList, &neighborlist1D[0],
                       neighborListCount * sizeof(int),
                       cudaMemcpyHostToDevice));
  gpuErrchk(cudaMemcpy(gpu_cellStartIndex, &cellStartIndex[0],
                       cellStartIndex.size() * sizeof(int),
                       cudaMemcpyHostToDevice));
  gpuErrchk(cudaMemcpy(vars->gpu_cellVector, &cellVector[0],
                       atomNumber * sizeof(int),
                       cudaMemcpyHostToDevice));
  cudaMemcpy(vars->gpu_x, currentCoords.x, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, currentCoords.y, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, currentCoords.z, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comx, currentCOM.x, molNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comy, currentCOM.y, molNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comz, currentCOM.z, molNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
             particleCharge.size() * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleKind, &particleKind[0],
             particleKind.size() * sizeof(int),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleMol, &particleMol[0],
             particleMol.size() * sizeof(int),
             cudaMemcpyHostToDevice);

  BoxInterForceGPU <<< blocksPerGrid, threadsPerBlock>>>(gpu_cellStartIndex,
                                                         vars->gpu_cellVector,
                                                         gpu_neighborList,
                                                         numberOfCells,
                                                         atomNumber,
                                                         vars->gpu_mapParticleToCell,
                                                         vars->gpu_x,
                                                         vars->gpu_y,
                                                         vars->gpu_z,
                                                         vars->gpu_comx,
                                                         vars->gpu_comy,
                                                         vars->gpu_comz,
                                                         boxAxes.GetAxis(box).x,
                                                         boxAxes.GetAxis(box).y,
                                                         boxAxes.GetAxis(box).z,
                                                         electrostatic,
                                                         gpu_particleCharge,
                                                         gpu_particleKind,
                                                         gpu_particleMol,
                                                         vars->gpu_rT11,
                                                         vars->gpu_rT12,
                                                         vars->gpu_rT13,
                                                         vars->gpu_rT22,
                                                         vars->gpu_rT23,
                                                         vars->gpu_rT33,
                                                         vars->gpu_vT11,
                                                         vars->gpu_vT12,
                                                         vars->gpu_vT13,
                                                         vars->gpu_vT22,
                                                         vars->gpu_vT23,
                                                         vars->gpu_vT33,
                                                         vars->gpu_sigmaSq,
                                                         vars->gpu_epsilon_Cn,
                                                         vars->gpu_n,
                                                         vars->gpu_VDW_Kind,
                                                         vars->gpu_isMartini,
                                                         vars->gpu_count,
                                                         vars->gpu_rCut,
                                                         vars->gpu_rCutCoulomb,
                                                         vars->gpu_rCutLow,
                                                         vars->gpu_rOn,
                                                         vars->gpu_alpha,
                                                         vars->gpu_ewald,
                                                         vars->gpu_diElectric_1,
                                                         vars->gpu_cell_x[box],
                                                         vars->gpu_cell_y[box],
                                                         vars->gpu_cell_z[box],
                                                         vars->gpu_Invcell_x[box],
                                                         vars->gpu_Invcell_y[box],
                                                         vars->gpu_Invcell_z[box],
                                                         vars->gpu_nonOrth,
                                                         sc_coul,
                                                         sc_sigma_6,
                                                         sc_alpha,
                                                         sc_power,
                                                         vars->gpu_rMin,
                                                         vars->gpu_rMaxSq,
                                                         vars->gpu_expConst,
                                                         box);
  checkLastErrorCUDA(__FILE__, __LINE__);
  cudaDeviceSynchronize();
  // ReduceSum // Virial of LJ
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT11,
                    gpu_final_value, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
                    threadsPerBlock);
  cudaMalloc(&d_temp_storage, temp_storage_bytes);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT11,
                    gpu_final_value, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
                    threadsPerBlock);
  cudaMemcpy(&vT11, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT12,
                    gpu_final_value, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
                    threadsPerBlock);
  cudaMemcpy(&vT12, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT13,
                    gpu_final_value, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
                    threadsPerBlock);
  cudaMemcpy(&vT13, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT22,
                    gpu_final_value, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
                    threadsPerBlock);
  cudaMemcpy(&vT22, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT23,
                    gpu_final_value, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
                    threadsPerBlock);
  cudaMemcpy(&vT23, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_vT33,
                    gpu_final_value, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
                    threadsPerBlock);
  cudaMemcpy(&vT33, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);

  if(electrostatic) {
    // ReduceSum // Virial of Coulomb
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT11,
                      gpu_final_value, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
                    threadsPerBlock);
    cudaMemcpy(&rT11, gpu_final_value, sizeof(double),
               cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT12,
                      gpu_final_value, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
                    threadsPerBlock);
    cudaMemcpy(&rT12, gpu_final_value, sizeof(double),
               cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT13,
                      gpu_final_value, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
                    threadsPerBlock);
    cudaMemcpy(&rT13, gpu_final_value, sizeof(double),
               cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT22,
                      gpu_final_value, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
                    threadsPerBlock);
    cudaMemcpy(&rT22, gpu_final_value, sizeof(double),
               cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT23,
                      gpu_final_value, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
                    threadsPerBlock);
    cudaMemcpy(&rT23, gpu_final_value, sizeof(double),
               cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT33,
                      gpu_final_value, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
                    threadsPerBlock);
    cudaMemcpy(&rT33, gpu_final_value, sizeof(double),
               cudaMemcpyDeviceToHost);
  }

  cudaFree(vars->gpu_rT11);
  cudaFree(vars->gpu_rT12);
  cudaFree(vars->gpu_rT13);
  cudaFree(vars->gpu_rT22);
  cudaFree(vars->gpu_rT23);
  cudaFree(vars->gpu_rT33);
  cudaFree(vars->gpu_vT11);
  cudaFree(vars->gpu_vT12);
  cudaFree(vars->gpu_vT13);
  cudaFree(vars->gpu_vT22);
  cudaFree(vars->gpu_vT23);
  cudaFree(vars->gpu_vT33);
  cudaFree(d_temp_storage);
  cudaFree(gpu_neighborList);
  cudaFree(gpu_cellStartIndex);
  cudaFree(gpu_particleKind);
  cudaFree(gpu_particleMol);
  cudaFree(gpu_particleCharge);
  cudaFree(gpu_final_value);
}

void CallBoxForceGPU(VariablesCUDA *vars,
                     vector<int> &cellVector,
                     vector<int> &cellStartIndex,
                     std::vector<std::vector<int> > &neighborList,
                     vector<int> &mapParticleToCell,
                     XYZArray const &coords,
                     BoxDimensions const &boxAxes,
                     bool electrostatic,
                     vector<double> &particleCharge,
                     vector<int> &particleKind,
                     vector<int> &particleMol,
                     double &REn,
                     double &LJEn,
                     double *aForcex,
                     double *aForcey,
                     double *aForcez,
                     double *mForcex,
                     double *mForcey,
                     double *mForcez,
                     int atomCount,
                     int molCount,
                     bool sc_coul,
                     double sc_sigma_6,
                     double sc_alpha,
                     uint sc_power,
                     uint const box)
{
  int atomNumber = coords.Count();
  int neighborListCount = neighborList.size() * NUMBER_OF_NEIGHBOR_CELL;
  int numberOfCells = neighborList.size();
  int *gpu_particleKind, *gpu_particleMol;
  int *gpu_neighborList, *gpu_cellStartIndex;
  int blocksPerGrid, threadsPerBlock;
  double *gpu_particleCharge;
  double *gpu_REn, *gpu_LJEn;
  double *gpu_final_REn, *gpu_final_LJEn;
  double cpu_final_REn, cpu_final_LJEn;

  // Run the kernel...
  threadsPerBlock = 256;
  blocksPerGrid = (int)(numberOfCells * NUMBER_OF_NEIGHBOR_CELL);

  // Convert neighbor list to 1D array
  std::vector<int> neighborlist1D(neighborListCount);
  for(int i=0; i<neighborList.size(); i++) {
    for(int j=0; j<NUMBER_OF_NEIGHBOR_CELL; j++) {
      neighborlist1D[i*NUMBER_OF_NEIGHBOR_CELL + j] = neighborList[i][j];
    }
  }

  cudaMemset(vars->gpu_aForcex, 0, atomCount * sizeof(double));
  cudaMemset(vars->gpu_aForcey, 0, atomCount * sizeof(double));
  cudaMemset(vars->gpu_aForcez, 0, atomCount * sizeof(double));
  cudaMemset(vars->gpu_mForcex, 0, molCount * sizeof(double));
  cudaMemset(vars->gpu_mForcey, 0, molCount * sizeof(double));
  cudaMemset(vars->gpu_mForcez, 0, molCount * sizeof(double));

  gpuErrchk(cudaMemcpy(vars->gpu_mapParticleToCell, &mapParticleToCell[0],
    atomNumber * sizeof(int), cudaMemcpyHostToDevice));
  gpuErrchk(cudaMalloc((void**) &gpu_neighborList, neighborListCount * sizeof(int)));
  gpuErrchk(cudaMalloc((void**) &gpu_cellStartIndex,
                       cellStartIndex.size() * sizeof(int)));
  cudaMalloc((void**) &gpu_particleCharge,
             particleCharge.size() * sizeof(double));
  cudaMalloc((void**) &gpu_particleKind, particleKind.size() * sizeof(int));
  cudaMalloc((void**) &gpu_particleMol, particleMol.size() * sizeof(int));
  cudaMalloc((void**) &gpu_REn, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
             threadsPerBlock * sizeof(double));
  cudaMalloc((void**) &gpu_LJEn, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
             threadsPerBlock * sizeof(double));
  cudaMalloc((void**) &gpu_final_REn, sizeof(double));
  cudaMalloc((void**) &gpu_final_LJEn, sizeof(double));

  // Copy necessary data to GPU
  gpuErrchk(cudaMemcpy(gpu_neighborList, &neighborlist1D[0],
                       neighborListCount * sizeof(int),
                       cudaMemcpyHostToDevice));
  gpuErrchk(cudaMemcpy(gpu_cellStartIndex, &cellStartIndex[0],
                       cellStartIndex.size() * sizeof(int),
                       cudaMemcpyHostToDevice));
  gpuErrchk(cudaMemcpy(vars->gpu_cellVector, &cellVector[0],
                       atomNumber * sizeof(int),
                       cudaMemcpyHostToDevice));
  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
             particleCharge.size() * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleKind, &particleKind[0],
             particleKind.size() * sizeof(int),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleMol, &particleMol[0], particleMol.size() * sizeof(int),
             cudaMemcpyHostToDevice);
  
  cudaMemcpy(vars->gpu_x, coords.x, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, coords.y, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, coords.z, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);

  checkLastErrorCUDA(__FILE__, __LINE__);
  BoxForceRealGPU <<< blocksPerGrid, threadsPerBlock, 0, vars->streams[1]>>>(gpu_cellStartIndex,
                                                    vars->gpu_cellVector,
                                                    gpu_neighborList,
                                                    numberOfCells,
                                                    atomNumber,
                                                    vars->gpu_mapParticleToCell,
                                                    vars->gpu_x,
                                                    vars->gpu_y,
                                                    vars->gpu_z,
                                                    boxAxes.GetAxis(box).x,
                                                    boxAxes.GetAxis(box).y,
                                                    boxAxes.GetAxis(box).z,
                                                    electrostatic,
                                                    gpu_particleCharge,
                                                    gpu_particleKind,
                                                    gpu_particleMol,
                                                    gpu_REn,
                                                    gpu_LJEn,
                                                    vars->gpu_sigmaSq,
                                                    vars->gpu_epsilon_Cn,
                                                    vars->gpu_n,
                                                    vars->gpu_VDW_Kind,
                                                    vars->gpu_isMartini,
                                                    vars->gpu_count,
                                                    vars->gpu_rCut,
                                                    vars->gpu_rCutCoulomb,
                                                    vars->gpu_rCutLow,
                                                    vars->gpu_rOn,
                                                    vars->gpu_alpha,
                                                    vars->gpu_ewald,
                                                    vars->gpu_diElectric_1,
                                                    vars->gpu_nonOrth,
                                                    vars->gpu_cell_x[box],
                                                    vars->gpu_cell_y[box],
                                                    vars->gpu_cell_z[box],
                                                    vars->gpu_Invcell_x[box],
                                                    vars->gpu_Invcell_y[box],
                                                    vars->gpu_Invcell_z[box],
                                                    vars->gpu_aForcex,
                                                    vars->gpu_aForcey,
                                                    vars->gpu_aForcez,
                                                    vars->gpu_mForcex,
                                                    vars->gpu_mForcey,
                                                    vars->gpu_mForcez,
                                                    sc_coul,
                                                    sc_sigma_6,
                                                    sc_alpha,
                                                    sc_power,
                                                    vars->gpu_rMin,
                                                    vars->gpu_rMaxSq,
                                                    vars->gpu_expConst,
                                                    box);
  BoxForceLJGPU <<< blocksPerGrid, threadsPerBlock, 0, vars->streams[0]>>>(gpu_cellStartIndex,
                                                    vars->gpu_cellVector,
                                                    gpu_neighborList,
                                                    numberOfCells,
                                                    atomNumber,
                                                    vars->gpu_mapParticleToCell,
                                                    vars->gpu_x,
                                                    vars->gpu_y,
                                                    vars->gpu_z,
                                                    boxAxes.GetAxis(box).x,
                                                    boxAxes.GetAxis(box).y,
                                                    boxAxes.GetAxis(box).z,
                                                    electrostatic,
                                                    gpu_particleCharge,
                                                    gpu_particleKind,
                                                    gpu_particleMol,
                                                    gpu_REn,
                                                    gpu_LJEn,
                                                    vars->gpu_sigmaSq,
                                                    vars->gpu_epsilon_Cn,
                                                    vars->gpu_n,
                                                    vars->gpu_VDW_Kind,
                                                    vars->gpu_isMartini,
                                                    vars->gpu_count,
                                                    vars->gpu_rCut,
                                                    vars->gpu_rCutCoulomb,
                                                    vars->gpu_rCutLow,
                                                    vars->gpu_rOn,
                                                    vars->gpu_alpha,
                                                    vars->gpu_ewald,
                                                    vars->gpu_diElectric_1,
                                                    vars->gpu_nonOrth,
                                                    vars->gpu_cell_x[box],
                                                    vars->gpu_cell_y[box],
                                                    vars->gpu_cell_z[box],
                                                    vars->gpu_Invcell_x[box],
                                                    vars->gpu_Invcell_y[box],
                                                    vars->gpu_Invcell_z[box],
                                                    vars->gpu_aForcex,
                                                    vars->gpu_aForcey,
                                                    vars->gpu_aForcez,
                                                    vars->gpu_mForcex,
                                                    vars->gpu_mForcey,
                                                    vars->gpu_mForcez,
                                                    sc_coul,
                                                    sc_sigma_6,
                                                    sc_alpha,
                                                    sc_power,
                                                    vars->gpu_rMin,
                                                    vars->gpu_rMaxSq,
                                                    vars->gpu_expConst,
                                                    box);


  cudaDeviceSynchronize();
  checkLastErrorCUDA(__FILE__, __LINE__);
  // ReduceSum
  void * d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_REn,
                    gpu_final_REn, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
                    threadsPerBlock);
  CubDebugExit(cudaMalloc(&d_temp_storage, temp_storage_bytes));
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_REn,
                    gpu_final_REn, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
                    threadsPerBlock);
  cudaFree(d_temp_storage);

  // LJ ReduceSum
  d_temp_storage = NULL;
  temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_LJEn,
                    gpu_final_LJEn, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
                    threadsPerBlock);
  CubDebugExit(cudaMalloc(&d_temp_storage, temp_storage_bytes));
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_LJEn,
                    gpu_final_LJEn, numberOfCells * NUMBER_OF_NEIGHBOR_CELL *
                    threadsPerBlock);
  cudaFree(d_temp_storage);
  // Copy back the result to CPU ! :)
  CubDebugExit(cudaMemcpy(&cpu_final_REn, gpu_final_REn, sizeof(double),
                          cudaMemcpyDeviceToHost));
  CubDebugExit(cudaMemcpy(&cpu_final_LJEn, gpu_final_LJEn, sizeof(double),
                          cudaMemcpyDeviceToHost));
  REn = cpu_final_REn;
  LJEn = cpu_final_LJEn;

  CubDebugExit(cudaMemcpy(aForcex, vars->gpu_aForcex,
                          sizeof(double) * atomCount,
                          cudaMemcpyDeviceToHost));
  CubDebugExit(cudaMemcpy(aForcey, vars->gpu_aForcey,
                          sizeof(double) * atomCount,
                          cudaMemcpyDeviceToHost));
  CubDebugExit(cudaMemcpy(aForcez, vars->gpu_aForcez,
                          sizeof(double) * atomCount,
                          cudaMemcpyDeviceToHost));
  CubDebugExit(cudaMemcpy(mForcex, vars->gpu_mForcex,
                          sizeof(double) * molCount,
                          cudaMemcpyDeviceToHost));
  CubDebugExit(cudaMemcpy(mForcey, vars->gpu_mForcey,
                          sizeof(double) * molCount,
                          cudaMemcpyDeviceToHost));
  CubDebugExit(cudaMemcpy(mForcez, vars->gpu_mForcez,
                          sizeof(double) * molCount,
                          cudaMemcpyDeviceToHost));

  cudaFree(gpu_particleCharge);
  cudaFree(gpu_particleKind);
  cudaFree(gpu_particleMol);
  cudaFree(gpu_REn);
  cudaFree(gpu_LJEn);
  cudaFree(gpu_final_REn);
  cudaFree(gpu_final_LJEn);
  cudaFree(gpu_neighborList);
  cudaFree(gpu_cellStartIndex);
  checkLastErrorCUDA(__FILE__, __LINE__);
}

void CallVirialReciprocalGPU(VariablesCUDA *vars,
                             XYZArray const &currentCoords,
                             XYZArray const &currentCOMDiff,
                             vector<double> &particleCharge,
                             double &rT11,
                             double &rT12,
                             double &rT13,
                             double &rT22,
                             double &rT23,
                             double &rT33,
                             uint imageSize,
                             double constVal,
                             uint box)
{
  int atomNumber = currentCoords.Count();
  int blocksPerGrid, threadsPerBlock;
  double *gpu_particleCharge;
  double *gpu_final_value;

  cudaMalloc((void**) &gpu_particleCharge,
             particleCharge.size() * sizeof(double));
  cudaMalloc((void**) &gpu_final_value, sizeof(double));

  cudaMemcpy(vars->gpu_x, currentCoords.x, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, currentCoords.y, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, currentCoords.z, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_dx, currentCOMDiff.x, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_dy, currentCOMDiff.y, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_dz, currentCOMDiff.z, atomNumber * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
             particleCharge.size() * sizeof(double),
             cudaMemcpyHostToDevice);

  // Run the kernel...
  threadsPerBlock = 256;
  blocksPerGrid = (int)(imageSize / threadsPerBlock) + 1;
  VirialReciprocalGPU <<< blocksPerGrid,
                      threadsPerBlock>>>(vars->gpu_x,
                                         vars->gpu_y,
                                         vars->gpu_z,
                                         vars->gpu_dx,
                                         vars->gpu_dy,
                                         vars->gpu_dz,
                                         vars->gpu_kxRef[box],
                                         vars->gpu_kyRef[box],
                                         vars->gpu_kzRef[box],
                                         vars->gpu_prefactRef[box],
                                         vars->gpu_hsqrRef[box],
                                         vars->gpu_sumRref[box],
                                         vars->gpu_sumIref[box],
                                         gpu_particleCharge,
                                         vars->gpu_rT11,
                                         vars->gpu_rT12,
                                         vars->gpu_rT13,
                                         vars->gpu_rT22,
                                         vars->gpu_rT23,
                                         vars->gpu_rT33,
                                         constVal,
                                         imageSize,
                                         atomNumber);
  cudaDeviceSynchronize();
  checkLastErrorCUDA(__FILE__, __LINE__);
  // ReduceSum // Virial of Reciprocal
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT11,
                    gpu_final_value, imageSize);
  cudaMalloc(&d_temp_storage, temp_storage_bytes);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT11,
                    gpu_final_value, imageSize);
  cudaMemcpy(&rT11, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT12,
                    gpu_final_value, imageSize);
  cudaMemcpy(&rT12, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT13,
                    gpu_final_value, imageSize);
  cudaMemcpy(&rT13, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT22,
                    gpu_final_value, imageSize);
  cudaMemcpy(&rT22, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT23,
                    gpu_final_value, imageSize);
  cudaMemcpy(&rT23, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, vars->gpu_rT33,
                    gpu_final_value, imageSize);
  cudaMemcpy(&rT33, gpu_final_value, sizeof(double),
             cudaMemcpyDeviceToHost);

  cudaFree(gpu_particleCharge);
  cudaFree(gpu_final_value);
  cudaFree(d_temp_storage);
}

__global__ void BoxInterForceGPU(int *gpu_cellStartIndex,
                                 int *gpu_cellVector,
                                 int *gpu_neighborList,
                                 int numberOfCells,
                                 int atomNumber,
                                 int *gpu_mapParticleToCell,
                                 double *gpu_x,
                                 double *gpu_y,
                                 double *gpu_z,
                                 double *gpu_comx,
                                 double *gpu_comy,
                                 double *gpu_comz,
                                 double xAxes,
                                 double yAxes,
                                 double zAxes,
                                 bool electrostatic,
                                 double *gpu_particleCharge,
                                 int *gpu_particleKind,
                                 int *gpu_particleMol,
                                 double *gpu_rT11,
                                 double *gpu_rT12,
                                 double *gpu_rT13,
                                 double *gpu_rT22,
                                 double *gpu_rT23,
                                 double *gpu_rT33,
                                 double *gpu_vT11,
                                 double *gpu_vT12,
                                 double *gpu_vT13,
                                 double *gpu_vT22,
                                 double *gpu_vT23,
                                 double *gpu_vT33,
                                 double *gpu_sigmaSq,
                                 double *gpu_epsilon_Cn,
                                 double *gpu_n,
                                 int *gpu_VDW_Kind,
                                 int *gpu_isMartini,
                                 int *gpu_count,
                                 double *gpu_rCut,
                                 double *gpu_rCutCoulomb,
                                 double *gpu_rCutLow,
                                 double *gpu_rOn,
                                 double *gpu_alpha,
                                 int *gpu_ewald,
                                 double *gpu_diElectric_1,
                                 double *gpu_cell_x,
                                 double *gpu_cell_y,
                                 double *gpu_cell_z,
                                 double *gpu_Invcell_x,
                                 double *gpu_Invcell_y,
                                 double *gpu_Invcell_z,
                                 int *gpu_nonOrth,
                                 bool sc_coul,
                                 double sc_sigma_6,
                                 double sc_alpha,
                                 uint sc_power,
                                 double *gpu_rMin,
                                 double *gpu_rMaxSq,
                                 double *gpu_expConst,
                                 int box)
{
  double distSq;
  double virX, virY, virZ;
  double pRF = 0.0, qi_qj, pVF = 0.0;
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  //tensors for VDW and real part of electrostatic
  gpu_vT11[threadID] = 0.0, gpu_vT22[threadID] = 0.0, gpu_vT33[threadID] = 0.0;
  gpu_rT11[threadID] = 0.0, gpu_rT22[threadID] = 0.0, gpu_rT33[threadID] = 0.0;
  // extra tensors reserved for later on
  gpu_vT12[threadID] = 0.0, gpu_vT13[threadID] = 0.0, gpu_vT23[threadID] = 0.0;
  gpu_rT12[threadID] = 0.0, gpu_rT13[threadID] = 0.0, gpu_rT23[threadID] = 0.0;
  double diff_comx, diff_comy, diff_comz;
  double cutoff = fmax(gpu_rCut[0], gpu_rCutCoulomb[box]);

  int currentCell = blockIdx.x / 27;
  int nCellIndex = blockIdx.x;
  int neighborCell = gpu_neighborList[nCellIndex];

  // calculate number of particles inside neighbor Cell
  int particlesInsideCurrentCell, particlesInsideNeighboringCells;
  int endIndex = neighborCell != numberOfCells - 1 ?
  gpu_cellStartIndex[neighborCell+1] : atomNumber;
  particlesInsideNeighboringCells = endIndex - gpu_cellStartIndex[neighborCell];

  // Calculate number of particles inside current Cell
  endIndex = currentCell != numberOfCells - 1 ?
  gpu_cellStartIndex[currentCell+1] : atomNumber;
  particlesInsideCurrentCell = endIndex - gpu_cellStartIndex[currentCell];

  // total number of pairs
  int numberOfPairs = particlesInsideCurrentCell * particlesInsideNeighboringCells;

  for(int pairIndex = threadIdx.x; pairIndex < numberOfPairs; pairIndex += blockDim.x) {
    int neighborParticleIndex = pairIndex / particlesInsideCurrentCell;
    int currentParticleIndex = pairIndex % particlesInsideCurrentCell;
    
    int currentParticle = gpu_cellVector[gpu_cellStartIndex[currentCell] + currentParticleIndex];
    int neighborParticle = gpu_cellVector[gpu_cellStartIndex[neighborCell] + neighborParticleIndex];

    if(currentParticle < neighborParticle) {
      // Check if they are within rcut
      if(InRcutGPU(distSq, virX, virY, virZ, gpu_x[currentParticle],
                  gpu_y[currentParticle], gpu_z[currentParticle],
                  gpu_x[neighborParticle], gpu_y[neighborParticle],
                  gpu_z[neighborParticle], xAxes, yAxes, zAxes, xAxes / 2.0,
                  yAxes / 2.0, zAxes / 2.0, cutoff, gpu_nonOrth[0],
                  gpu_cell_x, gpu_cell_y, gpu_cell_z, gpu_Invcell_x, gpu_Invcell_y,
                  gpu_Invcell_z)) {
        diff_comx = gpu_comx[gpu_particleMol[currentParticle]] -
                    gpu_comx[gpu_particleMol[neighborParticle]];
        diff_comy = gpu_comy[gpu_particleMol[currentParticle]] -
                    gpu_comy[gpu_particleMol[neighborParticle]];
        diff_comz = gpu_comz[gpu_particleMol[currentParticle]] -
                    gpu_comz[gpu_particleMol[neighborParticle]];

        diff_comx = MinImageSignedGPU(diff_comx, xAxes, xAxes / 2.0);
        diff_comy = MinImageSignedGPU(diff_comy, yAxes, yAxes / 2.0);
        diff_comz = MinImageSignedGPU(diff_comz, zAxes, zAxes / 2.0);

        if(electrostatic) {
          qi_qj = gpu_particleCharge[currentParticle] *
                  gpu_particleCharge[neighborParticle];
          pRF = CalcCoulombForceGPU(distSq, qi_qj, gpu_VDW_Kind[0], gpu_ewald[0],
                                    gpu_isMartini[0], gpu_alpha[box],
                                    gpu_rCutCoulomb[box], gpu_diElectric_1[0],
                                    gpu_sigmaSq, sc_coul, sc_sigma_6, sc_alpha,
                                    sc_power, gpu_count[0],
                                    gpu_particleKind[currentParticle],
                                    gpu_particleKind[neighborParticle]);

          gpu_rT11[threadID] += pRF * (virX * diff_comx);
          gpu_rT22[threadID] += pRF * (virY * diff_comy);
          gpu_rT33[threadID] += pRF * (virZ * diff_comz);

          //extra tensor calculations
          gpu_rT12[threadID] += pRF * (0.5 * (virX * diff_comy + virY * diff_comx));
          gpu_rT13[threadID] += pRF * (0.5 * (virX * diff_comz + virZ * diff_comx));
          gpu_rT23[threadID] += pRF * (0.5 * (virY * diff_comz + virZ * diff_comy));
        }

        pVF = CalcEnForceGPU(distSq, gpu_particleKind[currentParticle],
                            gpu_particleKind[neighborParticle],
                            gpu_sigmaSq, gpu_n, gpu_epsilon_Cn, gpu_rCut[0],
                            gpu_rOn[0], gpu_isMartini[0], gpu_VDW_Kind[0],
                            gpu_count[0], sc_sigma_6,
                            sc_alpha, sc_power, gpu_rMin, gpu_rMaxSq,
                            gpu_expConst);

        gpu_vT11[threadID] += pVF * (virX * diff_comx);
        gpu_vT22[threadID] += pVF * (virY * diff_comy);
        gpu_vT33[threadID] += pVF * (virZ * diff_comz);

        //extra tensor calculations
        gpu_vT12[threadID] += pVF * (0.5 * (virX * diff_comy + virY * diff_comx));
        gpu_vT13[threadID] += pVF * (0.5 * (virX * diff_comz + virZ * diff_comx));
        gpu_vT23[threadID] += pVF * (0.5 * (virY * diff_comz + virZ * diff_comy));
      }
    }
  }
}

__global__ void BoxForceLJGPU(int *gpu_cellStartIndex,
                            int *gpu_cellVector,
                            int *gpu_neighborList,
                            int numberOfCells,
                            int atomNumber,
                            int *gpu_mapParticleToCell,
                            double *gpu_x,
                            double *gpu_y,
                            double *gpu_z,
                            double xAxes,
                            double yAxes,
                            double zAxes,
                            bool electrostatic,
                            double *gpu_particleCharge,
                            int *gpu_particleKind,
                            int *gpu_particleMol,
                            double *gpu_REn,
                            double *gpu_LJEn,
                            double *gpu_sigmaSq,
                            double *gpu_epsilon_Cn,
                            double *gpu_n,
                            int *gpu_VDW_Kind,
                            int *gpu_isMartini,
                            int *gpu_count,
                            double *gpu_rCut,
                            double *gpu_rCutCoulomb,
                            double *gpu_rCutLow,
                            double *gpu_rOn,
                            double *gpu_alpha,
                            int *gpu_ewald,
                            double *gpu_diElectric_1,
                            int *gpu_nonOrth,
                            double *gpu_cell_x,
                            double *gpu_cell_y,
                            double *gpu_cell_z,
                            double *gpu_Invcell_x,
                            double *gpu_Invcell_y,
                            double *gpu_Invcell_z,
                            double *gpu_aForcex,
                            double *gpu_aForcey,
                            double *gpu_aForcez,
                            double *gpu_mForcex,
                            double *gpu_mForcey,
                            double *gpu_mForcez,
                            bool sc_coul,
                            double sc_sigma_6,
                            double sc_alpha,
                            uint sc_power,
                            double *gpu_rMin,
                            double *gpu_rMaxSq,
                            double *gpu_expConst,
                            int box)
{
  double distSq;
  double virX = 0.0, virY = 0.0, virZ = 0.0;
  double forceLJx = 0.0, forceLJy = 0.0, forceLJz = 0.0;
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  gpu_REn[threadID] = 0.0;
  gpu_LJEn[threadID] = 0.0;
  double cutoff = fmax(gpu_rCut[0], gpu_rCutCoulomb[box]);

  int currentCell = blockIdx.x / 27;
  int nCellIndex = blockIdx.x;
  int neighborCell = gpu_neighborList[nCellIndex];

  if(currentCell > neighborCell) return;

  // calculate number of particles inside neighbor Cell
  int particlesInsideCurrentCell, particlesInsideNeighboringCells;
  int endIndex = neighborCell != numberOfCells - 1 ?
  gpu_cellStartIndex[neighborCell+1] : atomNumber;
  particlesInsideNeighboringCells = endIndex - gpu_cellStartIndex[neighborCell];

  // Calculate number of particles inside current Cell
  endIndex = currentCell != numberOfCells - 1 ?
  gpu_cellStartIndex[currentCell+1] : atomNumber;
  particlesInsideCurrentCell = endIndex - gpu_cellStartIndex[currentCell];

  // total number of pairs
  int numberOfPairs = particlesInsideCurrentCell * particlesInsideNeighboringCells;

  for(int pairIndex = threadIdx.x; pairIndex < numberOfPairs; pairIndex += blockDim.x) {
    int neighborParticleIndex = pairIndex / particlesInsideCurrentCell;
    int currentParticleIndex = pairIndex % particlesInsideCurrentCell;
    
    int currentParticle = gpu_cellVector[gpu_cellStartIndex[currentCell] + currentParticleIndex];
    int neighborParticle = gpu_cellVector[gpu_cellStartIndex[neighborCell] + neighborParticleIndex];

    if(currentParticle != neighborParticle) {
      // Check if they are within rcut
      if(InRcutGPU(distSq, virX, virY, virZ, gpu_x[currentParticle],
        gpu_y[currentParticle], gpu_z[currentParticle],
        gpu_x[neighborParticle], gpu_y[neighborParticle],
        gpu_z[neighborParticle], xAxes, yAxes, zAxes, xAxes / 2.0,
        yAxes / 2.0, zAxes / 2.0, cutoff, gpu_nonOrth[0], gpu_cell_x,
        gpu_cell_y, gpu_cell_z, gpu_Invcell_x, gpu_Invcell_y,
        gpu_Invcell_z)) {
        // gpu_LJEn[threadID] += CalcEnGPU(distSq,
        //                       gpu_particleKind[currentParticle],
        //                       gpu_particleKind[neighborParticle],
        //                       gpu_sigmaSq, gpu_n, gpu_epsilon_Cn,
        //                       gpu_VDW_Kind[0], gpu_isMartini[0],
        //                       gpu_rCut[0], gpu_rOn[0], gpu_count[0],
        //                       sc_sigma_6, sc_alpha, sc_power, gpu_rMin,
        //                       gpu_rMaxSq, gpu_expConst);
        // double pVF = CalcEnForceGPU(distSq, gpu_particleKind[currentParticle],
        //                     gpu_particleKind[neighborParticle],
        //                     gpu_sigmaSq, gpu_n, gpu_epsilon_Cn,
        //                     gpu_rCut[0], gpu_rOn[0], gpu_isMartini[0],
        //                     gpu_VDW_Kind[0], gpu_count[0], sc_sigma_6,
        //                     sc_alpha, sc_power, gpu_rMin, gpu_rMaxSq,
        //                     gpu_expConst);
        int index = FlatIndexGPU(gpu_particleKind[currentParticle], gpu_particleKind[neighborParticle], gpu_count[0]);
        double rRat2 = gpu_sigmaSq[index] / distSq;
        double rRat4 = rRat2 * rRat2;
        double attract = rRat4 * rRat2;
        double repulse = pow(rRat2, gpu_n[index] / 2.0);
        gpu_LJEn[threadID] += gpu_epsilon_Cn[index] * (repulse - attract);
        double pVF = gpu_epsilon_Cn[index] * 6.0 *
          ((gpu_n[index] / 6.0) * repulse - attract) / distSq;

        forceLJx = virX * pVF;
        forceLJy = virY * pVF;
        forceLJz = virZ * pVF;

        atomicAdd(&gpu_aForcex[currentParticle], forceLJx);
        atomicAdd(&gpu_aForcey[currentParticle], forceLJy);
        atomicAdd(&gpu_aForcez[currentParticle], forceLJz);
        atomicAdd(&gpu_aForcex[neighborParticle], -1.0 * (forceLJx));
        atomicAdd(&gpu_aForcey[neighborParticle], -1.0 * (forceLJy));
        atomicAdd(&gpu_aForcez[neighborParticle], -1.0 * (forceLJz));

        atomicAdd(&gpu_mForcex[gpu_particleMol[currentParticle]],
          forceLJx);
        atomicAdd(&gpu_mForcey[gpu_particleMol[currentParticle]],
          forceLJy);
        atomicAdd(&gpu_mForcez[gpu_particleMol[currentParticle]],
          forceLJz);
        atomicAdd(&gpu_mForcex[gpu_particleMol[neighborParticle]],
          -1.0 * (forceLJx));
        atomicAdd(&gpu_mForcey[gpu_particleMol[neighborParticle]],
          -1.0 * (forceLJy));
        atomicAdd(&gpu_mForcez[gpu_particleMol[neighborParticle]],
          -1.0 * (forceLJz));
      }
    }
  }
}

__global__ void BoxForceRealGPU(int *gpu_cellStartIndex,
  int *gpu_cellVector,
  int *gpu_neighborList,
  int numberOfCells,
  int atomNumber,
  int *gpu_mapParticleToCell,
  double *gpu_x,
  double *gpu_y,
  double *gpu_z,
  double xAxes,
  double yAxes,
  double zAxes,
  bool electrostatic,
  double *gpu_particleCharge,
  int *gpu_particleKind,
  int *gpu_particleMol,
  double *gpu_REn,
  double *gpu_LJEn,
  double *gpu_sigmaSq,
  double *gpu_epsilon_Cn,
  double *gpu_n,
  int *gpu_VDW_Kind,
  int *gpu_isMartini,
  int *gpu_count,
  double *gpu_rCut,
  double *gpu_rCutCoulomb,
  double *gpu_rCutLow,
  double *gpu_rOn,
  double *gpu_alpha,
  int *gpu_ewald,
  double *gpu_diElectric_1,
  int *gpu_nonOrth,
  double *gpu_cell_x,
  double *gpu_cell_y,
  double *gpu_cell_z,
  double *gpu_Invcell_x,
  double *gpu_Invcell_y,
  double *gpu_Invcell_z,
  double *gpu_aForcex,
  double *gpu_aForcey,
  double *gpu_aForcez,
  double *gpu_mForcex,
  double *gpu_mForcey,
  double *gpu_mForcez,
  bool sc_coul,
  double sc_sigma_6,
  double sc_alpha,
  uint sc_power,
  double *gpu_rMin,
  double *gpu_rMaxSq,
  double *gpu_expConst,
  int box)
{
  double distSq;
  double qi_qj_fact;
  double qqFact = 167000.0;
  double virX = 0.0, virY = 0.0, virZ = 0.0;
  double forceRealx = 0.0, forceRealy = 0.0, forceRealz = 0.0;
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  gpu_REn[threadID] = 0.0;
  gpu_LJEn[threadID] = 0.0;
  double cutoff = fmax(gpu_rCut[0], gpu_rCutCoulomb[box]);

  int currentCell = blockIdx.x / 27;
  int nCellIndex = blockIdx.x;
  int neighborCell = gpu_neighborList[nCellIndex];

  if(currentCell > neighborCell) return;

  // calculate number of particles inside neighbor Cell
  int particlesInsideCurrentCell, particlesInsideNeighboringCells;
  int endIndex = neighborCell != numberOfCells - 1 ?
  gpu_cellStartIndex[neighborCell+1] : atomNumber;
  particlesInsideNeighboringCells = endIndex - gpu_cellStartIndex[neighborCell];

  // Calculate number of particles inside current Cell
  endIndex = currentCell != numberOfCells - 1 ?
  gpu_cellStartIndex[currentCell+1] : atomNumber;
  particlesInsideCurrentCell = endIndex - gpu_cellStartIndex[currentCell];

  // total number of pairs
  int numberOfPairs = particlesInsideCurrentCell * particlesInsideNeighboringCells;

  for(int pairIndex = threadIdx.x; pairIndex < numberOfPairs; pairIndex += blockDim.x) {
    int neighborParticleIndex = pairIndex / particlesInsideCurrentCell;
    int currentParticleIndex = pairIndex % particlesInsideCurrentCell;

    int currentParticle = gpu_cellVector[gpu_cellStartIndex[currentCell] + currentParticleIndex];
    int neighborParticle = gpu_cellVector[gpu_cellStartIndex[neighborCell] + neighborParticleIndex];

    if(currentParticle != neighborParticle) {
      // Check if they are within rcut
      if(InRcutGPU(distSq, virX, virY, virZ, gpu_x[currentParticle],
      gpu_y[currentParticle], gpu_z[currentParticle],
      gpu_x[neighborParticle], gpu_y[neighborParticle],
      gpu_z[neighborParticle], xAxes, yAxes, zAxes, xAxes / 2.0,
      yAxes / 2.0, zAxes / 2.0, cutoff, gpu_nonOrth[0], gpu_cell_x,
      gpu_cell_y, gpu_cell_z, gpu_Invcell_x, gpu_Invcell_y,
      gpu_Invcell_z)) {
      qi_qj_fact = gpu_particleCharge[currentParticle] *
      gpu_particleCharge[neighborParticle] * qqFact;
      gpu_REn[threadID] += CalcCoulombGPU(distSq,
                  gpu_particleKind[currentParticle],
                  gpu_particleKind[neighborParticle],
                  qi_qj_fact, gpu_rCutLow[0],
                  gpu_ewald[0], gpu_VDW_Kind[0],
                  gpu_alpha[box],
                  gpu_rCutCoulomb[box],
                  gpu_isMartini[0],
                  gpu_diElectric_1[0],
                  sc_coul,
                  sc_sigma_6,
                  sc_alpha,
                  sc_power,
                  gpu_sigmaSq,
                  gpu_count[0]);
      double coulombVir = CalcCoulombForceGPU(distSq, qi_qj_fact,
                        gpu_VDW_Kind[0], gpu_ewald[0],
                        gpu_isMartini[0],
                        gpu_alpha[box],
                        gpu_rCutCoulomb[box],
                        gpu_diElectric_1[0],
                        gpu_sigmaSq, sc_coul, sc_sigma_6,
                        sc_alpha, sc_power,
                        gpu_count[0],
                        gpu_particleKind[currentParticle],
                        gpu_particleKind[neighborParticle]);
      forceRealx = virX * coulombVir;
      forceRealy = virY * coulombVir;
      forceRealz = virZ * coulombVir;

      atomicAdd(&gpu_aForcex[currentParticle], forceRealx);
      atomicAdd(&gpu_aForcey[currentParticle], forceRealy);
      atomicAdd(&gpu_aForcez[currentParticle], forceRealz);
      atomicAdd(&gpu_aForcex[neighborParticle], -1.0 * (forceRealx));
      atomicAdd(&gpu_aForcey[neighborParticle], -1.0 * (forceRealy));
      atomicAdd(&gpu_aForcez[neighborParticle], -1.0 * (forceRealz));

      atomicAdd(&gpu_mForcex[gpu_particleMol[currentParticle]],
      forceRealx);
      atomicAdd(&gpu_mForcey[gpu_particleMol[currentParticle]],
      forceRealy);
      atomicAdd(&gpu_mForcez[gpu_particleMol[currentParticle]],
      forceRealz);
      atomicAdd(&gpu_mForcex[gpu_particleMol[neighborParticle]],
      -1.0 * (forceRealx));
      atomicAdd(&gpu_mForcey[gpu_particleMol[neighborParticle]],
      -1.0 * (forceRealy));
      atomicAdd(&gpu_mForcez[gpu_particleMol[neighborParticle]],
      -1.0 * (forceRealz));
      }
    }
  }
}

__global__ void VirialReciprocalGPU(double *gpu_x,
                                    double *gpu_y,
                                    double *gpu_z,
                                    double *gpu_comDx,
                                    double *gpu_comDy,
                                    double *gpu_comDz,
                                    double *gpu_kxRef,
                                    double *gpu_kyRef,
                                    double *gpu_kzRef,
                                    double *gpu_prefactRef,
                                    double *gpu_hsqrRef,
                                    double *gpu_sumRref,
                                    double *gpu_sumIref,
                                    double *gpu_particleCharge,
                                    double *gpu_rT11,
                                    double *gpu_rT12,
                                    double *gpu_rT13,
                                    double *gpu_rT22,
                                    double *gpu_rT23,
                                    double *gpu_rT33,
                                    double constVal,
                                    uint imageSize,
                                    uint atomNumber)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= imageSize)
    return;

  double factor, arg;
  int i;
  factor = gpu_prefactRef[threadID] * (gpu_sumRref[threadID] *
                                       gpu_sumRref[threadID] +
                                       gpu_sumIref[threadID] *
                                       gpu_sumIref[threadID]);
  gpu_rT11[threadID] = factor * (1.0 - 2.0 *
                                 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
                                 gpu_kxRef[threadID] * gpu_kxRef[threadID]);
  gpu_rT12[threadID] = factor * (-2.0 *
                                 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
                                 gpu_kxRef[threadID] * gpu_kyRef[threadID]);
  gpu_rT13[threadID] = factor * (-2.0 *
                                 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
                                 gpu_kxRef[threadID] * gpu_kzRef[threadID]);
  gpu_rT22[threadID] = factor * (1.0 - 2.0 *
                                 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
                                 gpu_kyRef[threadID] * gpu_kyRef[threadID]);
  gpu_rT23[threadID] = factor * (-2.0 *
                                 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
                                 gpu_kyRef[threadID] * gpu_kzRef[threadID]);
  gpu_rT33[threadID] = factor * (1.0 - 2.0 *
                                 (constVal + 1.0 / gpu_hsqrRef[threadID]) *
                                 gpu_kzRef[threadID] * gpu_kzRef[threadID]);

  //Intramolecular part
  for(i = 0; i < atomNumber; i++) {
    arg = DotProductGPU(gpu_kxRef[threadID], gpu_kyRef[threadID],
                        gpu_kzRef[threadID], gpu_x[i], gpu_y[i], gpu_z[i]);

    factor = gpu_prefactRef[threadID] * 2.0 *
             (gpu_sumIref[threadID] * cos(arg) - gpu_sumRref[threadID] * sin(arg)) *
             gpu_particleCharge[i];

    gpu_rT11[threadID] += factor * (gpu_kxRef[threadID] * gpu_comDx[i]);
    gpu_rT12[threadID] += factor * 0.5 * (gpu_kxRef[threadID] * gpu_comDy[i] +
                                          gpu_kyRef[threadID] * gpu_comDx[i]);
    gpu_rT13[threadID] += factor * 0.5 * (gpu_kxRef[threadID] * gpu_comDz[i] +
                                          gpu_kzRef[threadID] * gpu_comDx[i]);
    gpu_rT22[threadID] += factor * (gpu_kyRef[threadID] * gpu_comDy[i]);
    gpu_rT13[threadID] += factor * 0.5 * (gpu_kyRef[threadID] * gpu_comDz[i] +
                                          gpu_kzRef[threadID] * gpu_comDy[i]);
    gpu_rT33[threadID] += factor * (gpu_kzRef[threadID] * gpu_comDz[i]);
  }
}

__device__ double CalcEnForceGPU(double distSq, int kind1, int kind2,
                                 double *gpu_sigmaSq, double *gpu_n,
                                 double *gpu_epsilon_Cn, double gpu_rCut,
                                 double gpu_rOn, int gpu_isMartini,
                                 int gpu_VDW_Kind, int gpu_count,
                                 double sc_sigma_6,
                                 double sc_alpha, uint sc_power,
                                 double *gpu_rMin, double *gpu_rMaxSq,
                                 double *gpu_expConst)
{
  if((gpu_rCut * gpu_rCut) < distSq) {
    return 0.0;
  }

  int index = FlatIndexGPU(kind1, kind2, gpu_count);
  return CalcVirParticleGPU(distSq, index, gpu_sigmaSq, gpu_n,
                            gpu_epsilon_Cn, sc_sigma_6,
                            sc_alpha, sc_power);
}

//ElectroStatic Calculation
//**************************************************************//
__device__ double CalcCoulombVirParticleGPU(double distSq, double qi_qj,
    int gpu_ewald, double gpu_alpha,
    int index, double *gpu_sigmaSq,
    bool sc_coul, double sc_sigma_6,
    double sc_alpha, uint sc_power)
{
  if(gpu_ewald) {
    double dist = sqrt(distSq);
    double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
    double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
    double temp = 1.0 - erf(gpu_alpha * dist);
    return qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  } else {
    double dist = sqrt(distSq);
    double result = qi_qj / (distSq * dist);
    return result;
  }
}

//VDW Calculation
//*****************************************************************//
__device__ double CalcVirParticleGPU(double distSq, int index,
                                     double *gpu_sigmaSq, double *gpu_n,
                                     double *gpu_epsilon_Cn, double sc_sigma_6,
                                     double sc_alpha, uint sc_power)
{
  double rNeg2 = 1.0 / distSq;
  double rRat2 = gpu_sigmaSq[index] * rNeg2;
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index] / 2.0);
  return gpu_epsilon_Cn[index] * 6.0 *
         ((gpu_n[index] / 6.0) * repulse - attract) * rNeg2;
}

#endif
