
#include <hip/hip_runtime.h>
#ifdef GOMC_CUDA
#include "TransformParticlesCUDAKernel.cuh"
#include "CUDAMemoryManager.cuh"

#define MIN_FORCE 1E-12
#define MAX_FORCE 30

__device__ inline double randomGPU(unsigned int counter, unsigned int step, unsigned int seed) {
  RNG::ctr_type c = {{}};
  RNG::ukey_type uk = {{}};
  uk[0] = step;
  uk[1] = seed;
  RNG::key_type k = uk;
  c[0] = counter;
  RNG::ctr_type r = philox4x32(c, k);
  return (double)r[0] / UINT_MAX;
}

__device__ inline double WrapPBC(double &v, double ax) {
  if(v >= ax)
    v -= ax;
  else if(v < 0)
    v += ax;
  return v;
}

__device__ inline double UnwrapPBC(double &v, double ref, double ax, double halfax) {
  if(abs(ref - v) > halfax) {
    if(ref < halfax)
      v -= ax;
    else
      v += ax;
  }
  return v;
}

__device__ inline void ApplyRotation(double &x, double &y, double &z,
                                     double comx, double comy, double comz,
                                     double rotx, double roty, double rotz,
                                     double axx, double axy, double axz)
{
  double rotLen = sqrt(rotx * rotx + roty * roty + rotz * rotz);
  double axisx = rotx * (1.0 / rotLen);
  double axisy = roty * (1.0 / rotLen);
  double axisz = rotz * (1.0 / rotLen);
  double matrix[3][3], cross[3][3], tensor[3][3];

  // build cross
  cross[0][0] = 0.0;
  cross[0][1] = -axisz;
  cross[0][2] = axisy;

  cross[1][0] = axisz;
  cross[1][1] = 0.0;
  cross[1][2] = -axisx;

  cross[2][0] = -axisy;
  cross[2][1] = axisx;
  cross[2][2] = 0.0;

  // build tensor
  for(int i=0; i<3; i++) {
    tensor[0][i] = axisx;
    tensor[1][i] = axisy;
    tensor[2][i] = axisz;
  }
  for(int i=0; i<3; i++) {
    tensor[i][0] *= axisx;
    tensor[i][1] *= axisy;
    tensor[i][2] *= axisz;
  }

  // build matrix
  double c = cos(rotLen);
  for(int i=0; i<3; i++) {
    for(int j=0; j<3; j++) {
      matrix[i][j] = 0.0;
    }
    matrix[i][i] = c;
  }
  double s = sin(rotLen);
  for(int i=0; i<3; i++) {
    for(int j=0; j<3; j++) {
      matrix[i][j] += s * cross[i][j] + (1-c) * tensor[i][j];
    }
  }

  // unwrap molecule
  UnwrapPBC(x, comx, axx, axx/2.0);
  UnwrapPBC(y, comy, axy, axy/2.0);
  UnwrapPBC(z, comz, axz, axz/2.0);

  // move particle to zero
  x -= comx;
  y -= comy;
  z -= comz;

  // rotate
  double newx = matrix[0][0] * x + matrix[0][1] * y + matrix[0][2] * z;
  double newy = matrix[1][0] * x + matrix[1][1] * y + matrix[1][2] * z;
  double newz = matrix[2][0] * x + matrix[2][1] * y + matrix[2][2] * z;

  x = newx;
  y = newy;
  z = newz;

  // move back to com
  x += comx;
  y += comy;
  z += comz;

  // wrap again
  WrapPBC(x, axx);
  WrapPBC(y, axy);
  WrapPBC(z, axz);
}

void CallTranslateParticlesGPU(VariablesCUDA *vars,
                               std::vector<int> &isMoleculeInvolved,
                               double t_max,
                               double *mForcex,
                               double *mForcey,
                               double *mForcez,
                               unsigned int step,
                               unsigned int seed,
                               std::vector<int> particleMol,
                               int atomCount,
                               int molCount,
                               double xAxes,
                               double yAxes,
                               double zAxes,
                               XYZArray &newMolPos,
                               XYZArray &newCOMs,
                               double lambdaBETA,
                               XYZArray &t_k,
                               XYZArray &molForceRecRef)
{
  return;
  int *gpu_isMoleculeInvolved;
  int threadsPerBlock = 256;
  int blocksPerGrid = (int)(atomCount / threadsPerBlock) + 1;
  int *gpu_particleMol;

  CUMALLOC((void **) &gpu_isMoleculeInvolved,
           isMoleculeInvolved.size() * sizeof(int));
  CUMALLOC((void**) &gpu_particleMol, particleMol.size() * sizeof(int));

  cudaMemcpy(vars->gpu_mForcex, mForcex, molCount * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_mForcey, mForcey, molCount * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_mForcez, mForcez, molCount * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_mForceRecx, molForceRecRef.x, molCount * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_mForceRecy, molForceRecRef.y, molCount * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_mForceRecz, molForceRecRef.z, molCount * sizeof(double),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleMol, &particleMol[0], particleMol.size() * sizeof(int),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_isMoleculeInvolved, &isMoleculeInvolved[0],
             isMoleculeInvolved.size() * sizeof(int), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_x, newMolPos.x, atomCount * sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, newMolPos.y, atomCount * sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, newMolPos.z, atomCount * sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comx, newCOMs.x, molCount * sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comy, newCOMs.y, molCount * sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comz, newCOMs.z, molCount * sizeof(double), cudaMemcpyHostToDevice);

  checkLastErrorCUDA(__FILE__, __LINE__);
  TranslateParticlesKernel<<<blocksPerGrid, threadsPerBlock>>>(molCount,
                                                               t_max,
                                                               vars->gpu_mForcex,
                                                               vars->gpu_mForcey,
                                                               vars->gpu_mForcez,
                                                               step,
                                                               seed,
                                                               vars->gpu_x,
                                                               vars->gpu_y,
                                                               vars->gpu_z,
                                                               gpu_particleMol,
                                                               atomCount,
                                                               xAxes,
                                                               yAxes,
                                                               zAxes,
                                                               vars->gpu_comx,
                                                               vars->gpu_comy,
                                                               vars->gpu_comz,
                                                               lambdaBETA,
                                                               vars->gpu_t_k_x,
                                                               vars->gpu_t_k_y,
                                                               vars->gpu_t_k_z,
                                                               gpu_isMoleculeInvolved,
                                                               vars->gpu_mForceRecx,
                                                               vars->gpu_mForceRecy,
                                                               vars->gpu_mForceRecz);
  cudaDeviceSynchronize();
  checkLastErrorCUDA(__FILE__, __LINE__);
  
  cudaMemcpy(newMolPos.x, vars->gpu_x, atomCount * sizeof(double), cudaMemcpyDeviceToHost);
  cudaMemcpy(newMolPos.y, vars->gpu_y, atomCount * sizeof(double), cudaMemcpyDeviceToHost);
  cudaMemcpy(newMolPos.z, vars->gpu_z, atomCount * sizeof(double), cudaMemcpyDeviceToHost);
  cudaMemcpy(newCOMs.x, vars->gpu_comx, molCount * sizeof(double), cudaMemcpyDeviceToHost);
  cudaMemcpy(newCOMs.y, vars->gpu_comy, molCount * sizeof(double), cudaMemcpyDeviceToHost);
  cudaMemcpy(newCOMs.z, vars->gpu_comz, molCount * sizeof(double), cudaMemcpyDeviceToHost);
  cudaMemcpy(t_k.x, vars->gpu_t_k_x, molCount * sizeof(double), cudaMemcpyDeviceToHost);
  cudaMemcpy(t_k.y, vars->gpu_t_k_y, molCount * sizeof(double), cudaMemcpyDeviceToHost);
  cudaMemcpy(t_k.z, vars->gpu_t_k_z, molCount * sizeof(double), cudaMemcpyDeviceToHost);
  CUFREE(gpu_isMoleculeInvolved);
  CUFREE(gpu_particleMol);
  checkLastErrorCUDA(__FILE__, __LINE__);
}

void CallRotateParticlesGPU(VariablesCUDA *vars,
                            std::vector<int> &isMoleculeInvolved,
                            double r_max,
                            double *mTorquex,
                            double *mTorquey,
                            double *mTorquez,
                            unsigned int step,
                            unsigned int seed,
                            std::vector<int> particleMol,
                            int atomCount,
                            int molCount,
                            double xAxes,
                            double yAxes,
                            double zAxes,
                            XYZArray &newMolPos,
                            XYZArray &newCOMs,
                            double lambdaBETA,
                            XYZArray &r_k)
{
  return;
  int *gpu_isMoleculeInvolved;
  int threadsPerBlock = 256;
  int blocksPerGrid = (int)(atomCount / threadsPerBlock) + 1;
  int *gpu_particleMol;
  
  CUMALLOC((void **) &gpu_isMoleculeInvolved,
           isMoleculeInvolved.size() * sizeof(int));
  CUMALLOC((void**) &gpu_particleMol, particleMol.size() * sizeof(int));

  cudaMemcpy(vars->gpu_mTorquex, mTorquex, molCount * sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_mTorquey, mTorquey, molCount * sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_mTorquez, mTorquez, molCount * sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleMol, &particleMol[0], particleMol.size() * sizeof(int), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_x, newMolPos.x, atomCount * sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, newMolPos.y, atomCount * sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, newMolPos.z, atomCount * sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comx, newCOMs.x, molCount * sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comy, newCOMs.y, molCount * sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_comz, newCOMs.z, molCount * sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_isMoleculeInvolved, &isMoleculeInvolved[0],
    isMoleculeInvolved.size() * sizeof(int), cudaMemcpyHostToDevice);

  RotateParticlesKernel<<<blocksPerGrid, threadsPerBlock>>>(molCount,
                                                            r_max,
                                                            vars->gpu_mTorquex,
                                                            vars->gpu_mTorquey,
                                                            vars->gpu_mTorquez,
                                                            step,
                                                            seed,
                                                            vars->gpu_x,
                                                            vars->gpu_y,
                                                            vars->gpu_z,
                                                            gpu_particleMol,
                                                            atomCount,
                                                            xAxes,
                                                            yAxes,
                                                            zAxes,
                                                            vars->gpu_comx,
                                                            vars->gpu_comy,
                                                            vars->gpu_comz,
                                                            lambdaBETA,
                                                            vars->gpu_r_k_x,
                                                            vars->gpu_r_k_y,
                                                            vars->gpu_r_k_z,
                                                            gpu_isMoleculeInvolved);
  cudaDeviceSynchronize();
  checkLastErrorCUDA(__FILE__, __LINE__);
  
  cudaMemcpy(newMolPos.x, vars->gpu_x, atomCount * sizeof(double), cudaMemcpyDeviceToHost);
  cudaMemcpy(newMolPos.y, vars->gpu_y, atomCount * sizeof(double), cudaMemcpyDeviceToHost);
  cudaMemcpy(newMolPos.z, vars->gpu_z, atomCount * sizeof(double), cudaMemcpyDeviceToHost);
  cudaMemcpy(r_k.x, vars->gpu_r_k_x, molCount * sizeof(double), cudaMemcpyDeviceToHost);
  cudaMemcpy(r_k.y, vars->gpu_r_k_y, molCount * sizeof(double), cudaMemcpyDeviceToHost);
  cudaMemcpy(r_k.z, vars->gpu_r_k_z, molCount * sizeof(double), cudaMemcpyDeviceToHost);
  CUFREE(gpu_isMoleculeInvolved);
  CUFREE(gpu_particleMol);
  checkLastErrorCUDA(__FILE__, __LINE__);
}

__global__ void TranslateParticlesKernel(unsigned int numberOfMolecules,
                                         double t_max,
                                         double *molForcex,
                                         double *molForcey,
                                         double *molForcez,
                                         unsigned int step,
                                         unsigned int seed,
                                         double *gpu_x,
                                         double *gpu_y,
                                         double *gpu_z,
                                         int *gpu_particleMol,
                                         int atomCount,
                                         double xAxes,
                                         double yAxes,
                                         double zAxes,
                                         double *gpu_comx,
                                         double *gpu_comy,
                                         double *gpu_comz,
                                         double lambdaBETA,
                                         double *gpu_t_k_x,
                                         double *gpu_t_k_y,
                                         double *gpu_t_k_z,
                                         int *gpu_isMoleculeInvolved,
                                         double *gpu_mForceRecx,
                                         double *gpu_mForceRecy,
                                         double *gpu_mForceRecz)
{
  int atomNumber = blockIdx.x * blockDim.x + threadIdx.x;
  if(atomNumber >= atomCount) return;

  int molIndex = gpu_particleMol[atomNumber];
  if(!gpu_isMoleculeInvolved[molIndex]) return;
  bool updateMol = atomNumber == 0 || (gpu_particleMol[atomNumber] != gpu_particleMol[atomNumber-1]);

  // This section calculates the amount of shift
  double lbfx = (molForcex[molIndex] + gpu_mForceRecx[molIndex]) * lambdaBETA;
  double lbfy = (molForcey[molIndex] + gpu_mForceRecy[molIndex]) * lambdaBETA;
  double lbfz = (molForcez[molIndex] + gpu_mForceRecz[molIndex]) * lambdaBETA;
  double lbmaxx = lbfx * t_max;
  double lbmaxy = lbfy * t_max;
  double lbmaxz = lbfz * t_max;

  double shiftx, shifty, shiftz;

  if(abs(lbmaxx) > MIN_FORCE && abs(lbmaxx) < MAX_FORCE) {
    shiftx = log(exp(-1.0 * lbmaxx) + 2 * randomGPU(molIndex * 3, step, seed) * sinh(lbmaxx)) / lbfx;
  } else {
    double rr = randomGPU(molIndex * 3, step, seed) * 2.0 - 1.0;
    shiftx = t_max * rr;
  }

  if(abs(lbmaxy) > MIN_FORCE && abs(lbmaxy) < MAX_FORCE) {
    shifty = log(exp(-1.0 * lbmaxy) + 2 * randomGPU(molIndex * 3 + 1, step, seed) * sinh(lbmaxy)) / lbfy;
  } else {
    double rr = randomGPU(molIndex * 3 + 1, step, seed) * 2.0 - 1.0;
    shifty = t_max * rr;
  }

  if(abs(lbmaxz) > MIN_FORCE && abs(lbmaxz) < MAX_FORCE) {
    shiftz = log(exp(-1.0 * lbmaxz) + 2 * randomGPU(molIndex * 3 + 2, step, seed) * sinh(lbmaxz)) / lbfz;
  } else {
    double rr = randomGPU(molIndex * 3 + 2, step, seed) * 2.0 - 1.0;
    shiftz = t_max * rr;
  }

  // perform the shift on the coordinates
  gpu_x[atomNumber] += shiftx;
  gpu_y[atomNumber] += shifty;
  gpu_z[atomNumber] += shiftz;

  // rewrapping
  WrapPBC(gpu_x[atomNumber], xAxes);
  WrapPBC(gpu_y[atomNumber], yAxes);
  WrapPBC(gpu_z[atomNumber], zAxes);

  if(updateMol) {
    gpu_comx[molIndex] += shiftx;
    gpu_comy[molIndex] += shifty;
    gpu_comz[molIndex] += shiftz;

    WrapPBC(gpu_comx[molIndex], xAxes);
    WrapPBC(gpu_comy[molIndex], yAxes);
    WrapPBC(gpu_comz[molIndex], zAxes);

    gpu_t_k_x[molIndex] = shiftx;
    gpu_t_k_y[molIndex] = shifty;
    gpu_t_k_z[molIndex] = shiftz;
  }
}

__global__ void RotateParticlesKernel(unsigned int numberOfMolecules,
                                      double r_max,
                                      double *molTorquex,
                                      double *molTorquey,
                                      double *molTorquez,
                                      unsigned int step,
                                      unsigned int seed,
                                      double *gpu_x,
                                      double *gpu_y,
                                      double *gpu_z,
                                      int *gpu_particleMol,
                                      int atomCount,
                                      double xAxes,
                                      double yAxes,
                                      double zAxes,
                                      double *gpu_comx,
                                      double *gpu_comy,
                                      double *gpu_comz,
                                      double lambdaBETA,
                                      double *gpu_r_k_x,
                                      double *gpu_r_k_y,
                                      double *gpu_r_k_z,
                                      int *gpu_isMoleculeInvolved)
{
  int atomNumber = blockIdx.x * blockDim.x + threadIdx.x;
  if(atomNumber >= atomCount) return;
  int molIndex = gpu_particleMol[atomNumber];
  if(!gpu_isMoleculeInvolved[molIndex]) return;
  bool updateMol = atomNumber == 0 || (gpu_particleMol[atomNumber] != gpu_particleMol[atomNumber-1]);

  // This section calculates the amount of shift
  double lbtx = molTorquex[molIndex] * lambdaBETA;
  double lbty = molTorquey[molIndex] * lambdaBETA;
  double lbtz = molTorquez[molIndex] * lambdaBETA;
  double lbmaxx = lbtx * r_max;
  double lbmaxy = lbty * r_max;
  double lbmaxz = lbtz * r_max;

  double rotx, roty, rotz;

  if(abs(lbmaxx) > MIN_FORCE && abs(lbmaxx) < MAX_FORCE) {
    rotx = log(exp(-1.0 * lbmaxx) + 2 * randomGPU(molIndex * 3, step, seed) * sinh(lbmaxx)) / lbtx;
  } else {
    double rr = randomGPU(molIndex * 3, step, seed) * 2.0 - 1.0;
    rotx = r_max * rr;
  }

  if(abs(lbmaxy) > MIN_FORCE && abs(lbmaxy) < MAX_FORCE) {
    roty = log(exp(-1.0 * lbmaxy) + 2 * randomGPU(molIndex * 3 + 1, step, seed) * sinh(lbmaxy)) / lbty;
  } else {
    double rr = randomGPU(molIndex * 3 + 1, step, seed) * 2.0 - 1.0;
    roty = r_max * rr;
  }

  if(abs(lbmaxz) > MIN_FORCE && abs(lbmaxz) < MAX_FORCE) {
    rotz = log(exp(-1.0 * lbmaxz) + 2 * randomGPU(molIndex * 3 + 2, step, seed) * sinh(lbmaxz)) / lbtz;
  } else {
    double rr = randomGPU(molIndex * 3 + 2, step, seed) * 2.0 - 1.0;
    rotz = r_max * rr;
  }

  if(updateMol) {
    gpu_r_k_x[molIndex] = rotx;
    gpu_r_k_y[molIndex] = roty;
    gpu_r_k_z[molIndex] = rotz;
  }

  // perform the rot on the coordinates
  ApplyRotation(gpu_x[atomNumber], gpu_y[atomNumber], gpu_z[atomNumber],
                gpu_comx[molIndex], gpu_comy[molIndex], gpu_comz[molIndex],
                rotx, roty, rotz, xAxes, yAxes, zAxes);
}

#endif
